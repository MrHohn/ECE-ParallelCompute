#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "timing.h"
#include "util.h"

__global__ void upSweep(int* device_result, int length, int twod, int twod1) {
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * twod1;
    if ((index + twod1 - 1) >= length) {  // check boundary
        return;
    }
    device_result[index + twod1 - 1] += device_result[index + twod - 1];
}

__global__ void downSweep(int* device_result, int length, int twod, int twod1) {
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * twod1;
    if ((index + twod1 - 1) >= length) {  // check boudnary
        return;
    }
    int tmp = device_result[index + twod - 1];
    device_result[index + twod - 1] = device_result[index + twod1 - 1];
    device_result[index + twod1 - 1] += tmp;
}

void exclusive_scan(int* device_start, int length, int* device_result)
{
    const int roundUpLength = roundPowerTwo(length);
    int threadsPerBlock = 256;
    
    // upsweep phase
    for (int twod = 1; twod < roundUpLength; twod *= 2) {
        int twod1 = twod * 2;
        int blocksPerGrid 
            = (roundUpLength/twod1 + threadsPerBlock - 1) / threadsPerBlock;
        upSweep<<<blocksPerGrid, threadsPerBlock>>>(device_result, roundUpLength, twod, twod1);
    }

    // set last element to zero
    int zero = 0;
    hipMemcpy(&device_result[roundUpLength - 1], &zero, sizeof(int), hipMemcpyHostToDevice);

    // downsweep phase
    for (int twod = roundUpLength / 2; twod >= 1; twod /= 2) {
        int twod1 = twod * 2;
        int blocksPerGrid 
            = (roundUpLength/twod1 + threadsPerBlock - 1) / threadsPerBlock;
        downSweep<<<blocksPerGrid, threadsPerBlock>>>(device_result, roundUpLength, twod, twod1);
    }
}

double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;

    int rounded_length = roundPowerTwo(end - inarray);
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), 
               hipMemcpyHostToDevice);

    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), 
               hipMemcpyHostToDevice);

    // double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, end - inarray, device_result);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    // double endTime = CycleTimer::currentSeconds();
    // double overallDuration = endTime - startTime;
    
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);
    // free device memory
    hipFree(device_result);
    hipFree(device_input);
    
    return 0;
}

void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
