#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "timing.h"
#include "util.h"

__global__ void upSweep(int* device_result, int length, int twod, int twod1) {
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * twod1;
    if ((index + twod1 - 1) >= length) {  // check boundary
        return;
    }
    device_result[index + twod1 - 1] += device_result[index + twod - 1];
}

__global__ void downSweep(int* device_result, int length, int twod, int twod1) {
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * twod1;
    if ((index + twod1 - 1) >= length) {  // check boudnary
        return;
    }
    int tmp = device_result[index + twod - 1];
    device_result[index + twod - 1] = device_result[index + twod1 - 1];
    device_result[index + twod1 - 1] += tmp;
}

void exclusive_scan(int* device_start, int len, int* device_result)
{
    const int roundUpLength = roundPowerTwo(len);
    int threadsPerBlock = 256;
    
    // upsweep phase
    for (int twod = 1; twod < roundUpLength; twod *= 2) {
        int twod1 = twod * 2;
        int blocksPerGrid 
            = (roundUpLength/twod1 + threadsPerBlock - 1) / threadsPerBlock;
        upSweep<<<blocksPerGrid, threadsPerBlock>>>(device_result, roundUpLength, twod, twod1);
    }

    // set last element to zero
    int zero = 0;
    hipMemcpy(&device_result[roundUpLength - 1], &zero, sizeof(int), hipMemcpyHostToDevice);

    // downsweep phase
    for (int twod = roundUpLength / 2; twod >= 1; twod /= 2) {
        int twod1 = twod * 2;
        int blocksPerGrid 
            = (roundUpLength/twod1 + threadsPerBlock - 1) / threadsPerBlock;
        downSweep<<<blocksPerGrid, threadsPerBlock>>>(device_result, roundUpLength, twod, twod1);
    }
}

double exclusive_scan_parallel(int* nums, int len, int* output)
{
    int* device_result;
    int* device_input;

    int roundedLen = roundPowerTwo(len);

    hipMalloc((void **)&device_result, roundedLen * sizeof(int));
    hipMalloc((void **)&device_input, roundedLen * sizeof(int));
    hipMemcpy(device_input, nums, len * sizeof(int), 
               hipMemcpyHostToDevice);
    hipMemcpy(device_result, nums, len * sizeof(int), 
               hipMemcpyHostToDevice);

    // double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, len, device_result);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    // double endTime = CycleTimer::currentSeconds();
    // double overallDuration = endTime - startTime;
    
    hipMemcpy(output, device_result, len * sizeof(int),
               hipMemcpyDeviceToHost);
    // free device memory
    hipFree(device_result);
    hipFree(device_input);

    return 0;
}

void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
