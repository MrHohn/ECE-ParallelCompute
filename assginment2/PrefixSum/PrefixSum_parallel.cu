#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "timing.h"
#include "util.h"

// this method is cited from CMU's assignment description
// http://15418.courses.cs.cmu.edu/spring2015/article/4
__global__ void buildUpTree(int* device_result, int len, int starter, int interval)
{
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * interval;
    // return if out of range
    if (index + interval > len)
    {  
        return;
    }
    device_result[index + interval - 1] += device_result[index + starter - 1];
}

__global__ void buildDownTree(int* device_result, int len, int starter, int interval)
{
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * interval;
    // return if out of range
    if (index + interval > len)
    {  
        return;
    }
    int temp = device_result[index + starter - 1];
    device_result[index + starter - 1] = device_result[index + interval - 1];
    device_result[index + interval - 1] += temp;
}

void launch_scan(int roundedLen, int* device_result)
{
    int threadsPerBlock = 256;
    
    // build up tree
    for (int starter = 1; starter < roundedLen; starter *= 2)
    {
        int interval = starter * 2;
        // plus (threadsPerBlock - 1) is actually ceiling the block number
        int numBlocks = (roundedLen / interval + threadsPerBlock - 1) / threadsPerBlock;
        buildUpTree<<<numBlocks, threadsPerBlock>>>(device_result, roundedLen, starter, interval);
    }

    // set last element to zero
    int zero = 0;
    hipMemcpy(&device_result[roundedLen - 1], &zero, sizeof(int), hipMemcpyHostToDevice);

    // build down tree
    for (int starter = roundedLen / 2; starter >= 1; starter /= 2)
    {
        int interval = starter * 2;
        // plus (threadsPerBlock - 1) is actually ceiling the block number
        int numBlocks = (roundedLen / interval + threadsPerBlock - 1) / threadsPerBlock;
        buildDownTree<<<numBlocks, threadsPerBlock>>>(device_result, roundedLen, starter, interval);
    }
}


void exclusive_scan_parallel(int* nums, int len, int* output, double& time_cost)
{
    int* device_result;

    int roundedLen = roundPowerTwo(len);

    // Allocate space on GPU and copy inputs into it
    hipMalloc((void **)&device_result, roundedLen * sizeof(int));
    hipMemcpy(device_result, nums, len * sizeof(int), hipMemcpyHostToDevice);

    // Start to do GPU computing
    reset_and_start_timer();

    // Since in-place algorithm are used, we did not allocate device_input
    launch_scan(roundedLen, device_result);

    // Wait for all instances to finished
    hipDeviceSynchronize();

    // Finished GPU computing
    time_cost = get_elapsed_mcycles();
    
    // Copy back the result
    hipMemcpy(output, device_result, len * sizeof(int), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_result);
}
