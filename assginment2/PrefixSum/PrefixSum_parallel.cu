#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "timing.h"
#include "util.h"

// this method is cited from CMU's assignment description
// http://15418.courses.cs.cmu.edu/spring2015/article/4
__global__ void buildUpTree(int* device_output, int len, int starter, int interval)
{
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * interval;
    // return if out of range
    if (index + interval > len)
    {  
        return;
    }
    device_output[index + interval - 1] += device_output[index + starter - 1];
}

__global__ void buildDownTree(int* device_output, int len, int starter, int interval)
{
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * interval;
    // return if out of range
    if (index + interval > len)
    {  
        return;
    }
    int temp = device_output[index + starter - 1];
    device_output[index + starter - 1] = device_output[index + interval - 1];
    device_output[index + interval - 1] += temp;
}

void launch_scan(int roundedLen, int* device_output, int threadsPerBlock)
{   
    // build up tree
    for (int starter = 1; starter < roundedLen; starter *= 2)
    {
        int interval = starter * 2;
        // plus (threadsPerBlock - 1) is actually ceiling the block number
        int numBlocks = (roundedLen / interval + threadsPerBlock - 1) / threadsPerBlock;
        buildUpTree<<<numBlocks, threadsPerBlock>>>(device_output, roundedLen, starter, interval);
    }

    // set last element to zero
    int zero = 0;
    hipMemcpy(&device_output[roundedLen - 1], &zero, sizeof(int), hipMemcpyHostToDevice);

    // build down tree
    for (int starter = roundedLen / 2; starter >= 1; starter /= 2)
    {
        int interval = starter * 2;
        // plus (threadsPerBlock - 1) is actually ceiling the block number
        int numBlocks = (roundedLen / interval + threadsPerBlock - 1) / threadsPerBlock;
        buildDownTree<<<numBlocks, threadsPerBlock>>>(device_output, roundedLen, starter, interval);
    }
}

void exclusive_scan_parallel(int* nums, int len, int* output, double& time_cost)
{
    int* device_output;

    int roundedLen = roundPowerTwo(len);

    // Allocate space on GPU and copy inputs into it
    hipMalloc((void **)&device_output, roundedLen * sizeof(int));
    hipMemcpy(device_output, nums, len * sizeof(int), hipMemcpyHostToDevice);

    // Start to do GPU computing
    reset_and_start_timer();

    // Since in-place algorithm is used, we did not allocate device_input
    launch_scan(roundedLen, device_output, 256);

    // Wait for all instances to finished
    hipDeviceSynchronize();

    // Finished GPU computing
    time_cost = get_elapsed_mcycles();
    
    // Copy back the result
    hipMemcpy(output, device_output, len * sizeof(int), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_output);
}

__global__ void setRepeat(int* device_input, int* device_outputBTemp, int* device_outputCTemp, int len)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // return if out of range
    if (index + 1 >= len)
    {
        return;
    }

    // set repeated bits
    if (device_input[index] == device_input[index + 1])
    {
        device_outputBTemp[index] = 1;
        device_outputCTemp[index] = 0;
    }
    else
    {
        device_outputBTemp[index] = 0;
        device_outputCTemp[index] = 1;
    }
}

__global__ void getRepeat(int* nums, int* device_outputB, int* device_outputC, int* device_outputBTemp, int* device_outputCTemp, int len)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // check last bit
    if (index + 1 == len)
    {
        int curC = device_outputCTemp[index];
        if (curC != device_outputCTemp[index - 1])
        {
            device_outputC[curC] = nums[index];
        }
        return;
    }
    // return if out of range
    if (index + 1 > len)
    {
        return;
    }

    int curB = device_outputBTemp[index];
    int curC = device_outputCTemp[index];
    if (curB < device_outputBTemp[index + 1])
    {
        device_outputB[curB] = index;
    }
    if (curC < device_outputCTemp[index + 1])
    {
        device_outputC[curC] = nums[index];
    }
}

int launch_find(int len, int roundedLen, int* device_input, int* device_outputB, int* device_outputC, int* device_outputBTemp, int* device_outputCTemp, int threadsPerBlock)
{
    // call setRepeat function first
    int numBlocks = (len + threadsPerBlock - 1) / threadsPerBlock;
    setRepeat<<<numBlocks, threadsPerBlock>>>(device_input, device_outputBTemp, device_outputCTemp, len);

    // call exclusive scan again to add up previous output
    launch_scan(roundedLen, device_outputBTemp, threadsPerBlock);
    launch_scan(roundedLen, device_outputCTemp, threadsPerBlock);

    // get number of repeats
    int repeat_count;
    hipMemcpy(&repeat_count, &device_outputBTemp[len - 1], sizeof(int), hipMemcpyDeviceToHost);

    // call getRepeat function now
    getRepeat<<<numBlocks, threadsPerBlock>>>(device_input, device_outputB, device_outputC, device_outputBTemp, device_outputCTemp, len);

    return repeat_count;
}

int find_repeats_parallel(int* nums, int len, int* outputB, int* outputC, double& time_cost)
{
    int* device_input;
    int* device_outputB;
    int* device_outputBTemp;
    int* device_outputC;
    int* device_outputCTemp;

    int roundedLen = roundPowerTwo(len);

    // Allocate space on GPU and copy inputs into it
    hipMalloc((void **)&device_input, len * sizeof(int));
    hipMalloc((void **)&device_outputB, len * sizeof(int));
    hipMalloc((void **)&device_outputC, len * sizeof(int));
    hipMalloc((void **)&device_outputBTemp, roundedLen * sizeof(int));
    hipMalloc((void **)&device_outputCTemp, roundedLen * sizeof(int));
    hipMemcpy(device_input, nums, len * sizeof(int), hipMemcpyHostToDevice);
    
    // Start to do GPU computing
    reset_and_start_timer();

    int repeat_count = launch_find(len, roundedLen, device_input, device_outputB, device_outputC, device_outputBTemp, device_outputCTemp, 256);

    // Wait for all instances to finished
    hipDeviceSynchronize();

    // Finished GPU computing
    time_cost = get_elapsed_mcycles();
    
    // Copy back the result
    hipMemcpy(outputB, device_outputB, repeat_count * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(outputC, device_outputC, (len - repeat_count) * sizeof(int), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_input);
    hipFree(device_outputB);
    hipFree(device_outputC);
    hipFree(device_outputBTemp);
    hipFree(device_outputCTemp);

    return repeat_count;
}
